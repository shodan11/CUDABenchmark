
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>

__global__ void test() {}



__global__ void foo_kernel(int *d_flag)
{
	int tid = blockIdx.x*blockDim.x+threadIdx.x;
	int flag = 1;
	
}

void foo()
{
	int *d_flag;
	int flag = 1;
	at_quick_exit(test);
	hipMalloc((void**)&d_flag, sizeof(int));
    hipMemcpy((void *)d_flag, (const void *)&flag, sizeof(int), hipMemcpyHostToDevice);
    foo_kernel<<<1,10>>>(d_flag);
    hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
}

int main() {
  at_quick_exit(foo);
  quick_exit(0);
}

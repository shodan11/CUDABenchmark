#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>


__global__ void fun(int *z){     
    int x = 5;
    int a[-x];
    *z = a[-x];
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}

//编译错误; gcc:0;clang:segmentation fault: 11;nvcc: error: expression must have a constant value；

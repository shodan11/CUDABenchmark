#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
#include <string.h>


__global__ void fun(int *z){  
        int x = 0;
        (int)(&x) & -7;
        *z = (int)(&x) & -7;
        printf("%d\n", (int)(&x) & -7);
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}

//编译通过；gcc:4;clang:4;nvcc:0;Unspecified value or behavior.C11 sec. 5.1.2.2.3:1;

#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <math.h>

__global__ void fun(float x, float y, float *z)
//_gloable_void fun(float x, float y, float *z)
//一行代码三处低级错误；
{

  *z = pow(x, y) ;
 
}

int main(void)
{
  float z;
  float *dev_z;
  hipMalloc((void**)&dev_z, sizeof(float));
  fun<<<1,1>>>(10.0, 39.0, dev_z);
  hipMemcpy(&z, dev_z, sizeof(float), hipMemcpyDeviceToHost);
  printf("%f\n", z);//printf（"%f\n", z）;括号的错误
  hipFree(dev_z);
  return 0;
  
}

//输出结果为inf;

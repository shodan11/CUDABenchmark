#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

struct str {int *x;};

__global__ void fun(int *y){
     int x = 5;
     int *p = &x;
     (union str)p;
     //printf("%p\n", *y);
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost); 
  hipFree(dev_y);  
}

//编译错误类型：j044d.cu(12): error: tag kind of union is incompatible with declaration of class "str"


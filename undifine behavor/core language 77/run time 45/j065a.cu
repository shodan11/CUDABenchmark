#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>

__global__ void fun(int *y){
      volatile int x = 5;
	*(int*)&x;
        *y = *(int*)&x;
        printf("%d\n", *y);
      //return 0;
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过; gcc:5;nvcc: 5;clang: 编译的时候出现警告：expression result unused [-wunused-value] *(int*)&x;

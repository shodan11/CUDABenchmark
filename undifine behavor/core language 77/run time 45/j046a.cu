#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void fun(int *y){
     int a;
     &a+2;
     printf("%p\n", &a+2);
     
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过 0x3fffc78；

#include <stdlib.h>
#include <iostream>

#include "hip/hip_runtime.h"
#include <stdio.h>


__global__ void test(int *z) 
{
	char x, y;
	if (x==y)
	{
		*z=0;
	}
	printf("%d\n", *z);	
} 

int main(void)
{
	int z;
	int *dev_z;
	hipMalloc((void**)&dev_z, sizeof(int));
	test<<<1,1>>>(dev_z);
	hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyHostToDevice);
	printf("%d\n",z);
	hipFree(dev_z);
	return 0;
}                                                                                   
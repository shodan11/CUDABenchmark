
#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void fun(int *z){ 
        int x = 5;
        float* p = (float*)&x;
        *p;
        printf("%f\n", *p);
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}



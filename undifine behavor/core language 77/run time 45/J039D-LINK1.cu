#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
int foo();

__global__ void foo_kernel(int x, int *y)
{
  
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  foo_kernel<<<1,1>>>(5,dev_y);
  hipMemcpy(&y,dev_y, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n",y);
  hipFree(dev_y);
  
}
//0


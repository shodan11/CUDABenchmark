#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>

__global__ void fun(int *y){
      const int x = 0;
      unsigned char* p = (unsigned char*)&x;
      *p = 5;
      *y = *p;
      printf("%d\n", *y);
      //return 0;
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过; gcc:5;nvcc: 5;

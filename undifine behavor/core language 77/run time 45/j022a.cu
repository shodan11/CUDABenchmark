#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
#include <string.h>


__global__ void fun(int *z){ 
        register int a[5] = {0};
        a[0];
        printf("%d\n", a[0]);
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);  
  hipFree(dev_z);
  return 0;
  
}



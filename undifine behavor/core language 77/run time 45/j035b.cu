#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void fun(int *z){ 
        int x = 0;
      //return (x = 1) / (x = 1);
      printf("%d\n", (x = 1) / (x = 1));
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}
#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>

__global__ void fun(int *y){
      int a;
      int b;
      
      if (&a >= &b) {
            *y=1;
      }
      else *y=0;
      printf("%d\n", *y);
     
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过; gcc:0;nvcc:1;

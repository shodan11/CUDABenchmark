#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void fun(int *y){
     int a[5] = {1, 2, 3, 4, 5};
     *y = *(&a[5]);
     printf("%d\n", *(&a[5]));
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过 1；

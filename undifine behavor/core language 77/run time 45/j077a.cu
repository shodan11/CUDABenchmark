#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
#include <stdalign.h>

__global__ void f_kernel(int *d_flag){
        int tid = blockIdx.x*blockDim.x+threadIdx.x;
        int flag = 1;

        if(threadIdx.x == 0)
             *d_flag = 0;
}
  int f(int a[static 5]){
  
  int *d_flag;
  int flag = 1;
  hipMalloc((void**)&d_flag, sizeof(int));
  hipMemcpy(d_flag,0,sizeof(int), hipMemcpyHostToDevice);
  f_kernel<<<1,1>>>(d_flag);
  hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
}

int main(void)
{
  int a[2];
  //printf("%d\n", f(a));
  return f(a);
}
        
 
//编译错误;nvcc:(17): error: expected an expression;把static去掉就没问题；

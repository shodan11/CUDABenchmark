#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef struct test_struct {
  void *               r;
  int                  y;
  double               x;
  long                 z;
  char                 str[348];
  struct test_struct * next;
  struct test_struct * prev;

} test_struct;

__global__ void fun(int *z){  
  void *        voidp   = NULL;
  char          zeroes[sizeof(test_struct)];

  memset(zeroes, 0, sizeof(zeroes));

  const void *s1 = &voidp;
  const void *s2 = zeroes;
  size_t n = sizeof(voidp);
  const unsigned char *us1 = (const unsigned char *) s1;
  const unsigned char *us2 = (const unsigned char *) s2;
  int k;
  while (n-- != 0) {
        if (*us1 != *us2) {
               if(*us1 < *us2){k = -1;}
               else k = 1;
              
        }
        printf("%d\n", k);
        us1++;
        us2++;
  }
    printf("%d\n", 100); 
  //return 100;
      // printf("%d\n", (int)(&x) & -7);
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}

//编译通过；

#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>

__global__ void fun(int *y){
      struct big_type {
      int arr[32];
};
      union u {
      struct s1 { char c; struct big_type bt1; } sub1;
      struct s2 { long long x; struct big_type bt2; } sub2;
} obj;
      obj.sub2.bt2 = obj.sub1.bt1;
      printf("%d\n", obj.sub2.bt2);
      printf("%d\n", obj.sub1.bt1);
     
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过; nvcc: 0 0;gcc:1526236216   2147483637;

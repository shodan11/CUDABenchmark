#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <limits.h>
#include <stddef.h>

__global__ void fun(int *y){
     if (sizeof(ptrdiff_t) == sizeof(int)) {
            unsigned char *ptr0 = malloc(((unsigned)INT_MAX) + 1);

            unsigned char *ptr1 = ptr0 + (unsigned)INT_MAX + 1;

            ptr1 - ptr0;
            *y = ptr1 - ptr0;
            printf("%ld\n", *y);
            }
            //return 0;                 
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost); 
  hipFree(dev_y);
  return 0;
  
}

//j050a.cu(12): error: a value of type "void *" cannot be used to initialize an entity of type "unsigned char *"

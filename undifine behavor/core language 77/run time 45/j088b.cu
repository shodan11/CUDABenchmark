#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
#include <stdalign.h>

__global__ void f_kernel(int *d_flag){
        int tid = blockIdx.x*blockDim.x+threadIdx.x;
        int flag = 1;

        if(threadIdx.x == 0)
             *d_flag = 0;
}
  int f(void){
  
  int *d_flag;
  int flag = 1;
  hipMalloc((void**)&d_flag, sizeof(int));
  hipMemcpy(d_flag,0,sizeof(int), hipMemcpyHostToDevice);
  f_kernel<<<1,1>>>(d_flag);
  hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
}

int main(void)
{
  f() + 0;
  printf("%d\n", f() + 0);
  //return 0;
}
        
 
//编译通过；

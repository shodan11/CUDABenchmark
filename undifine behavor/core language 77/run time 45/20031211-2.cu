#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

struct a
{
  unsigned int bitfield : 3;
};

__device__ void test(int tid, int z, int *d_flag)
{
  if ( tid == 0) {
    if (z != 3)
      *d_flag = 0;
  }
}

__global__ void foo_kernel(int *d_flag)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  struct a b;
  b.bitfield = 131;

  test(tid, b.bitfield, d_flag);
}

void foo()
{
  int *d_flag;
  int flag = 1;
  hipMalloc((void**)&d_flag, sizeof(int));
  hipMemcpy((void *)d_flag, (const void *)&flag, sizeof(int), hipMemcpyHostToDevice);
  foo_kernel<<<1,1>>>(d_flag);
  hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);

  if (flag == 0) {
    printf("aborting\n");
    abort();
  }
}

int main()
{
  foo();
  //printf("%d\n", foo());
  //exit (0);
}
#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>


__global__ void fun(int *z){     
     int x = 5;
     int y = 5;
     int * __restrict__ p = &x;
     int * __restrict__ q = &y;
printf("%d\n", *p);
     p = q;
printf("%d\n", *p);
      *z = *p;     
      printf("%d\n", *z);    
      //return 0;            
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}

//编译通过; gcc:5;nvcc: 5；

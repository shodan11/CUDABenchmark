#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>

__global__ void f_kernel(int* __restrict__ a, int* __restrict__ b, int *d_flag){
        int tid = blockIdx.x*blockDim.x+threadIdx.x;
        int flag = 1;
        *a = 1;
	*b = 1;
        if(threadIdx.x == 0)
             *d_flag = 0;
}
int f(int* __restrict__ a, int* __restrict__ b)
{
  int *d_flag;
  int flag = 1;
  hipMalloc((void**)&d_flag, sizeof(int));
  hipMemcpy(d_flag, a, sizeof(int), hipMemcpyHostToDevice);
  f_kernel<<<1,1>>>(a,b,d_flag);
  hipMemcpy(&flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);
}

int main(void)
{
  int a = 5;
  printf("%d\n", f(&a, &a));
  return f(&a, &a);
}
        
 
//编译通过; gcc:5;nvcc: 77；

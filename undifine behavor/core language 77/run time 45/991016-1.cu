#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
typedef int T0;
typedef long T1;
typedef long long T2;


__global__ void doit(int sel, int n, void *p, int *y){
  
  T0 * const p0 = p;
  T1 * const p1 = p;
  T2 * const p2 = p;
  switch (sel)
    {
    case 0:
      do 
    *p0 += *p0;
      while (--n);
      return *p0 == 0;

    case 1:
      do 
   *p1 += *p1;
      while (--n);
      return *p1 == 0;

    case 2:
      do 
    *p2 += *p2;
      while (--n);
      return *p2 == 0;

    default:
      abort ();
    }
*y = p;

}



int main()
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  doit<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  T0 v0; T1 v1; T2 v2;

  v0 = 1; doit(0, 5, &v0);
  printf("%d\n", v0);
  v1 = 1; doit(1, 5, &v1);
  printf("%d\n", v1);
  v2 = 1; doit(2, 5, &v2);
  printf("%d\n", v2);
  //hipFree(dev_y);
  //return 0;
  
}

//编译未通过; gcc:32;nvcc: ；

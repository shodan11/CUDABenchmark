#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
#include <string.h>


__global__ void fun(int *z){ 
        int x;
        x + 1;
        int y = x + 1;
        printf("%d\n", y);
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}


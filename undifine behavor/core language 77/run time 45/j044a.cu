#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

__global__ void fun(double *y){
     int x = 5;
     int *p = &x;
     *y = (double)p;
     printf("%d\n", y);
             
}

int main(void)
{
  double y;
  double *dev_y;
  hipMalloc((void**)&dev_y, sizeof(double));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(double), hipMemcpyDeviceToHost); 
  hipFree(dev_y);  
}

//编译错误类型：044a.cu(10): error: invalid type conversion

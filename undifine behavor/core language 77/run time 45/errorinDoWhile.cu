#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>


__global__ void fun(int *z){ 
      int x = 1;

      do {
            x = x - 1;
      } while (x / x);
      printf("%d\n", x);
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}
#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>


__global__ void fun(int *z){  

    printf("%d\n", (long long int)(7));
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}

//编译通过；gcc:7;nvcc:7;

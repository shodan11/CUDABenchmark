#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
#include <string.h>


__global__ void fun(int *z){ 
      int buf[5] = {0};
      for (int i = 5; buf[i] == 0; i++) {
            printf("%d\n", i);
      }
      printf("%d\n", buf[5]);
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}
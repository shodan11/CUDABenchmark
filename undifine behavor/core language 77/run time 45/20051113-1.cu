#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>
extern void *memset(void *, int, size_t);
typedef struct
{
  short a;  
  unsigned short b;
  unsigned short c;
  unsigned long long Count;
  long long Count2;
} __attribute__((packed)) Struct1;

typedef struct
{
  short a;
  unsigned short b;
  unsigned short c;
  unsigned long long d;
  long long e;
  long long f;
} __attribute__((packed)) Struct2;

typedef union
{
  Struct1 a;
  Struct2 b;
} Union;

typedef struct
{
  int Count;
  Union List[];
} __attribute__((packed)) Struct3;
unsigned long long Sum (Struct3 *instrs) __attribute__((noinline));

__global__ void Sum_kernel (Struct3 *instrs,unsigned long long count,unsigned long long *y)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long Count = 0;
  int i;
  for (i = 0; i < instrs->Count; i++) {
    count += instrs->List[i].a.Count;
    *y = count;
  }
}
unsigned long long Sum(Struct3 *instrs)
{
  unsigned long long *y;
  int flag = 1;
  hipMalloc((void**)&y, sizeof(unsigned long long));
  hipMemcpy((void *)y, (const void *)&flag, sizeof(unsigned long long), hipMemcpyHostToDevice);
  Sum_kernel<<<1,1>>>(instrs, y);
  hipMemcpy(&flag, y, sizeof(unsigned long long), hipMemcpyDeviceToHost);

}


int main()
{
  Struct3 *p = malloc (sizeof (int) + 3 * sizeof(Union));
  memset(p, 0, sizeof(int) + 3*sizeof(Union));
  p->Count = 3;
  p->List[0].a.Count = 555;
  p->List[1].a.Count = 999;
  p->List[2].a.Count = 0x101010101ULL;
  p->List[0].a.Count2 = 555;
  p->List[1].a.Count2 = 999;
  p->List[2].a.Count2 = 0x101010101LL;
  if (Sum(p) != 555 + 999 + 0x101010101ULL)
    abort(); 
}



//nvcc: allowing all exceptions is incompatible with previous function "memset"
///usr/local/cuda-7.0/bin/..//include/common_functions.h(72): here
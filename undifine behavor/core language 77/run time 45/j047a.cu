#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void fun(int *y){
     int a = 5;
     *(&a+1);
     printf("%d\n", *(&a+1));
     
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过 5；

#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

__global__ void f_kernel(int *y)
{
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
}
int f(int z){
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  f_kernel<<<1,1>>>(dev_y);
  hipMemcpy(&z, dev_y, sizeof(int), hipMemcpyDeviceToHost);
}

int main(void)
{
  int (*z)(double) = (int (*)(double))&f;
  //return z(5);
  printf("%d\n", z(5));
}



#include <stdlib.h>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>

#include <stddef.h>
#include <limits.h>
#include <string.h>


__global__ void fun(int *z){ 
        char a[5] = {1, 2, 3, 4, 5};
        printf("%d\n", *(int*)(&a[4]));
}

int main(void)
{
  int z;
  int *dev_z;
  hipMalloc((void**)&dev_z, sizeof(int));
  fun<<<1,1>>>(dev_z);
  hipMemcpy(&z, dev_z, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_z);
  return 0;
  
}

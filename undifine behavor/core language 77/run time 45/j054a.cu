#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <stddef.h>
#include <limits.h>

__global__ void fun(int *y){
      long long *p = malloc(sizeof(long long));

      *p = 5;

      *p = *(short*)p;
      
      printf("%d\n", *(short*)p);
     
             
}

int main(void)
{
  int y;
  int *dev_y;
  hipMalloc((void**)&dev_y, sizeof(int));
  fun<<<1,1>>>(dev_y);
  hipMemcpy(&y, dev_y, sizeof(int), hipMemcpyDeviceToHost);
  
  hipFree(dev_y);
  return 0;
  
}

//编译通过; gcc:5;nvcc: error: a value of type "void *" cannot be used to initialize an entity of type "long long *";
